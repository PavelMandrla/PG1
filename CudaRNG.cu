#include "hip/hip_runtime.h"
#include "CudaRNG.h"
#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include ""
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <Windows.h>
#include <omp.h>

//http://ianfinlayson.net/class/cpsc425/notes/cuda-random

__global__ void init(unsigned int seed, hiprandState_t* states) {
	hiprand_init(seed,
		blockIdx.x,
		0,
		&states[blockIdx.x]);
}

/* this GPU kernel takes an array of states, and an array of ints, and puts a random int into each */
__global__ void randoms(hiprandState_t* states, unsigned int* numbers) {
	numbers[blockIdx.x] = hiprand(&states[blockIdx.x]);
}


void CudaRNG::getRandomNums(const int bufferSize, int * buffer) {
	#pragma omp critical GPUAccess
	{
		hiprandState_t* states;
		hipMalloc((void**)&states, bufferSize * sizeof(hiprandState_t));
		init <<<bufferSize, 1 >> > (GetTickCount(), states);

		unsigned int* gpu_nums;
		hipMalloc((void**)&gpu_nums, bufferSize * sizeof(unsigned int));

		randoms <<<bufferSize, 1 >> > (states, gpu_nums);
		hipMemcpy(buffer, gpu_nums, bufferSize * sizeof(unsigned int), hipMemcpyDeviceToHost);

		hipFree(states);
		hipFree(gpu_nums);
	}
}