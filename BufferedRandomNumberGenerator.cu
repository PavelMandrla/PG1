#include "hip/hip_runtime.h"
#include "BufferedRandomNumberGenerator.h"

#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include "hip/hip_runtime.h"
#include ""

__global__ void generateRandomInts(unsigned int seed, int* result) {
	hiprandState_t state;
	hiprand_init(seed, /* the seed controls the sequence of random values that are produced */
		blockIdx.x, /* the sequence number is only important with multiple cores */
		0, /* the offset is how much extra we advance in the sequence for each call, can be 0 */
		&state);
	*result = hiprand(&state);

}
/*
void CudaRNG::fillBuffer(int * buffer, int bufferSize) {
	//BufferedRandomNumberGenerator::mtx.lock();

	int* gpu_x;
	hipMalloc((void**)&gpu_x, sizeof(int));
	//generateRandomInts<<<1, 1 >>>(123, gpu_x);

	int x;
	hipMemcpy(&x, gpu_x, sizeof(int), hipMemcpyDeviceToHost);
	printf("Random number = %d.\n", x);
	hipFree(gpu_x);


	//BufferedRandomNumberGenerator::mtx.unlock();
}*/